
#include <hip/hip_runtime.h>
#include <stdio.h>

#include<sys/time.h>
#define MAX 500

__global__ void dkernel(int *res,int n,int *res_size,int v2) {
    // Initialize result
    res[0] = 1;
//    int res_size = 1;
 
    for (int x=2; x<=n; x++){
        int carry = 0;  // Initialize carry
        for (int i=0; i<*res_size; i++)
        {
            int prod = res[i] * x + carry;
            res[i] = prod % 10; 
            carry  = prod/10;    
        }
     
        while (carry)
        {
            res[*res_size] = carry%10;
            carry = carry/10;
            (*res_size)++;
        }
    
    }
     int flag=0;  
     for(int g=1;g<=v2;g++){
    	if(g%2==0)
    		flag++;
    	else
    		flag--;
    	__syncthreads();//barrier here
 
    }
}


void soln(int *res,int n,int *res_size){
    // Initialize result
    res[0] = 1;
//    int res_size = 1;
 	int x;
    for (x=2; x<=n; x++){
        int carry = 0,i;  // Initialize carry
        for (i=0; i<*res_size; i++)
        {
            int prod = res[i] * x + carry;
            res[i] = prod % 10; 
            carry  = prod/10;    
        }
     
        while (carry)
        {
            res[*res_size] = carry%10;
            carry = carry/10;
            (*res_size)++;
        }
    
    }
}


int main(int nn, char *str[]) {
    int *res, *hres;
    int res_size=1;
    
    
    hipMalloc(&res, MAX * sizeof(int));
    hres = (int*)malloc(MAX*sizeof(int));
//    soln(hres,N,&res_size);
	for(int N=50;N>=1;N-=2){
		for(int v2=10000;v2>=1;v2-=300){
		struct timeval  tv1, tv2;
		gettimeofday(&tv1, NULL);
		for(int i=0;i<100000;i++)
   	 		dkernel<<<1,1>>>(res,N,&res_size,v2);
    	hipMemcpy(hres, res, MAX * sizeof(int), hipMemcpyDeviceToHost);
    //    cout << "Factorial of given number is \n";
    
     	hipDeviceSynchronize();
   		gettimeofday(&tv2, NULL);
			printf ("%d %d %f\n",N,v2,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
    }}
 //   int i;
    //for (i=res_size-1; i>=0; i--)
    //    printf("%d",hres[i]);
   // printf("\n");
    return 0;
}

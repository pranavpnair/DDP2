
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<sys/time.h>
#define MAX 500


__device__ void dkernel(int *res,int n,int *res_size) {
    // Initialize result
    res[0] = 1;
//    int res_size = 1;
 
    for (int x=2; x<=n; x++){
        int carry = 0;  // Initialize carry
        for (int i=0; i<*res_size; i++)
        {
            int prod = res[i] * x + carry;
            res[i] = prod % 10; 
            carry  = prod/10;    
        }
     
        while (carry)
        {
            res[*res_size] = carry%10;
            carry = carry/10;
            (*res_size)++;
        }
    
    }
}


void soln(int *res,int n,int *res_size){
    // Initialize result
    res[0] = 1;
//    int res_size = 1;
 	int x;
    for (x=2; x<=n; x++){
        int carry = 0,i;  // Initialize carry
        for (i=0; i<*res_size; i++)
        {
            int prod = res[i] * x + carry;
            res[i] = prod % 10; 
            carry  = prod/10;    
        }
     
        while (carry)
        {
            res[*res_size] = carry%10;
            carry = carry/10;
            (*res_size)++;
        }
    
    }
}

__global__ void five(int * res, int val2,int *res_size,int val1){
	
	int idx = threadIdx.x;
	
	switch(idx%val1){
	    case 0:dkernel(res,val2,res_size);;break;
	    case 1:dkernel(res,val2,res_size);;break;
	    case 2:dkernel(res,val2,res_size);;break;
	    case 3:dkernel(res,val2,res_size);;break;
	    case 4:dkernel(res,val2,res_size);;break;
	    case 5:dkernel(res,val2,res_size);;break;
	    case 6:dkernel(res,val2,res_size);;break;
	    case 7:dkernel(res,val2,res_size);;break;
	    case 8:dkernel(res,val2,res_size);;break;
	    case 9:dkernel(res,val2,res_size);;break;
	    case 10:dkernel(res,val2,res_size);;break;
	    case 11:dkernel(res,val2,res_size);;break;
	    case 12:dkernel(res,val2,res_size);;break;
	    case 13:dkernel(res,val2,res_size);;break;
	    case 14:dkernel(res,val2,res_size);;break;
	    case 15:dkernel(res,val2,res_size);;break;
	    case 16:dkernel(res,val2,res_size);;break;
	    case 17:dkernel(res,val2,res_size);;break;
	    case 18:dkernel(res,val2,res_size);;break;
	    case 19:dkernel(res,val2,res_size);;break;
	    case 20:dkernel(res,val2,res_size);;break;
	    case 21:dkernel(res,val2,res_size);;break;
	    case 22:dkernel(res,val2,res_size);;break;
	    case 23:dkernel(res,val2,res_size);;break;
	    case 24:dkernel(res,val2,res_size);;break;
	    case 25:dkernel(res,val2,res_size);;break;
	    case 26:dkernel(res,val2,res_size);;break;
	    case 27:dkernel(res,val2,res_size);;break;
	    case 28:dkernel(res,val2,res_size);;break;
	    case 29:dkernel(res,val2,res_size);;break;
	    case 30:dkernel(res,val2,res_size);;break;
	    case 31:dkernel(res,val2,res_size);;break;
	}
	__syncthreads();
}




int main(int nn, char *str[]) {
    int *res, *hres;
    int res_size=1;
    
    
    hipMalloc(&res, MAX * sizeof(int));
    hres = (int*)malloc(MAX*sizeof(int));
//    soln(hres,N,&res_size);
	for(int v1=32;v1>=1;v1--){
		for(int v2=32;v2>=1;v2--){
		struct timeval  tv1, tv2;
			gettimeofday(&tv1, NULL);
	for(int i=0;i<100000;i++)
    	five<<<1,32>>>(res,v2,&res_size,v1);
    hipMemcpy(hres, res, MAX * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    gettimeofday(&tv2, NULL);
			printf ("%d %d %f\n",v1,v2,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
    //    cout << "Factorial of given number is \n";
 //   for (i=res_size-1; i>=0; i--)
 //       printf("%d",hres[i]);
  //  printf("\n");
  
  
    }}
    return 0;
}

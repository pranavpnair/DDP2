
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<sys/time.h>


__device__ void dkernel(float *vector, unsigned vectorsize,int N) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id<vectorsize)   
        vector[id]+=N;  
}

__global__ void five(float * d_in, int val1,int val2){
	
	int idx = threadIdx.x;
	
	switch(idx%val1){
	    case 0: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 1: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 2: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 3: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 4: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 5: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 6: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 7: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 8: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 9: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 10: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 11: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 12: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 13: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 14: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 15: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 16: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 17: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 18: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 19: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 20: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 21: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 22: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 23: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 24: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 25: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 26: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 27: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 28: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 29: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 30: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	    case 31: for(int i=0;i<val2;i++)dkernel(d_in,1024,val2);break;
	}
	__syncthreads();
}




int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 1024;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	
	
	for(int v1=32;v1>=1;v1--){
		for(int v2=1000;v2>=1;v2-=100){
		
			struct timeval  tv1, tv2;
			gettimeofday(&tv1, NULL);

			for(long long int j=0;j<1000000;j++){
				five<<<1, ARRAY_SIZE>>>(d_in,v1,v2);
				hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
				hipDeviceSynchronize();
			}
			
			gettimeofday(&tv2, NULL);
			printf ("%d %d %f\n",v1,v2,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
		}
	}
	// copy back the result array to the CPU
	

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
		



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>


__global__ void coal(int * d_out, int * d_in,int val1,int val2){
	int f;
	int totalSum=1;
	for(int i=0;i<1024;i+=val2)
	{
		f= d_in[i];
		d_out[i]=f*f;
		
	if (threadIdx.x == 0) totalSum = 0;
  	  __syncthreads();

    for(int i=0;i<val1;i++)
    	atomicAdd(&totalSum, d_out[i]);
    __syncthreads();
	}
}


int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 1024;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

	// generate the input array on the host
	int h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = int(i);
	}
	int h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	int * d_in;
	int * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	


	
	for(int v1=32;v1>=1;v1--){
		for(int v2=10000;v2>=100;v2-=300){
		
			struct timeval  tv1, tv2;
			gettimeofday(&tv1, NULL);
		//	cudaEventRecord(start);
			for(long long int j=0;j<1000000;j++){
				coal<<<100, ARRAY_SIZE>>>(d_out, d_in,v1,v2);
				hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
				hipDeviceSynchronize();
			}
			
		//	cudaEventSynchronize(stop);
		//	cudaEventElapsedTime(&time, start, stop);
			gettimeofday(&tv2, NULL);
			printf ("%d %d %f\n",v1,v2,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
		}
	}


	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
		

#include <stdio.h>
#include <hip/hip_runtime.h>
#include<sys/time.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize,int N,int v2) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id<vectorsize)   
        vector[id]+=N;
    int flag=0;  
     for(int g=1;g<=v2;g++){
    	if(g%2==0)
    		flag++;
    	else
    		flag--;
    	__syncthreads();//barrier here
 
    }
}

#define BLOCKSIZE 1024


int main(int nn, char *str[]) {
    unsigned long long N = 1024;
    unsigned *vector, *hvector;
    unsigned vec[N];
    for (int i = 0; i < N; i++) {
		vec[i] = i;
	}
	
    hipMalloc(&vector, N * sizeof(unsigned));
    hipMemcpy(vector, vec, N * sizeof(unsigned), hipMemcpyHostToDevice);
    hvector = (unsigned *)malloc(N * sizeof(unsigned));
    unsigned nblocks = ceil((float)N / BLOCKSIZE);
//    printf("nblocks = %d\n", nblocks);

//here we run  the kernel in a loop which runs 1024 times.
	for(int v1=10000;v1>=1;v1-=1000){
	for(int j=1000000;j>=1;j-=10000) {
		struct timeval  tv1, tv2;
		gettimeofday(&tv1, NULL);
    	for(int i=0;i<j;i++){
    	    dkernel<<<nblocks, BLOCKSIZE>>>(vector, N,i,v1);
    	     hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);}
    	gettimeofday(&tv2, NULL);
	
		printf ("%d %d %f\n",j,v1,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
		hipDeviceSynchronize();
   	}
   	}
   
    /*for (unsigned ii = 0; ii < N; ++ii) {
    printf("%4d ", hvector[ii]);
    }*/
    return 0;
}

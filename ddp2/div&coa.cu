
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<sys/time.h>



__device__ void coal(float * d_out, float * d_in,int val2){
	float f;
	for(int i=0;i<1024;i+=val2)
	{
		f= d_in[i];
		d_out[i]=f*f;
	}
}

__global__ void three(float * d_out, float * d_in,int val1,int val2){
	
	int idx = threadIdx.x;
	
	switch(idx%val1){
	    case 0: coal(d_out,d_in,val2);break;
	    case 1: coal(d_out,d_in,val2);break;
	    case 2: coal(d_out,d_in,val2);break;
	    case 3: coal(d_out,d_in,val2);break;
	    case 4: coal(d_out,d_in,val2);break;
	    case 5: coal(d_out,d_in,val2);break;
	    case 6: coal(d_out,d_in,val2);break;
	    case 7: coal(d_out,d_in,val2);break;
	    case 8: coal(d_out,d_in,val2);break;
	    case 9: coal(d_out,d_in,val2);break;
	    case 10: coal(d_out,d_in,val2);break;
	    case 11: coal(d_out,d_in,val2);break;
	    case 12: coal(d_out,d_in,val2);break;
	    case 13: coal(d_out,d_in,val2);break;
	    case 14: coal(d_out,d_in,val2);break;
	    case 15: coal(d_out,d_in,val2);break;
	    case 16: coal(d_out,d_in,val2);break;
	    case 17: coal(d_out,d_in,val2);break;
	    case 18: coal(d_out,d_in,val2);break;
	    case 19: coal(d_out,d_in,val2);break;
	    case 20: coal(d_out,d_in,val2);break;
	    case 21: coal(d_out,d_in,val2);break;
	    case 22: coal(d_out,d_in,val2);break;
	    case 23: coal(d_out,d_in,val2);break;
	    case 24: coal(d_out,d_in,val2);break;
	    case 25: coal(d_out,d_in,val2);break;
	    case 26: coal(d_out,d_in,val2);break;
	    case 27: coal(d_out,d_in,val2);break;
	    case 28: coal(d_out,d_in,val2);break;
	    case 29: coal(d_out,d_in,val2);break;
	    case 30: coal(d_out,d_in,val2);break;
	    case 31: coal(d_out,d_in,val2);break;
	}
	__syncthreads();
}



int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 1024;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	


	
	for(int v1=32;v1>=1;v1--){
		for(int v2=32;v2>=1;v2--){
		
			struct timeval  tv1, tv2;
			gettimeofday(&tv1, NULL);
		//	cudaEventRecord(start);
			for(long long int j=0;j<1000000;j++){
				three<<<100, ARRAY_SIZE>>>(d_out, d_in,v1,v2);
				hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
				hipDeviceSynchronize();
			}
			
		//	cudaEventSynchronize(stop);
		//	cudaEventElapsedTime(&time, start, stop);
			gettimeofday(&tv2, NULL);
			printf ("%d %d %f\n",v1,v2,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
		}
	}


	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
		


#include <hip/hip_runtime.h>
#include <stdio.h>
#include<sys/time.h>



__device__ void barrier(float *vector,int i) {
 unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    vector[id] = id;
    int flag=0;
    for(int g=1;g<=i;g++){
    	if(g%2==0)
    		flag++;
    	else
    		flag--;
    	__syncthreads();//barrier here
 
    }
}

__global__ void five(float * d_out, float * d_in,int val1,int val2){
	
	int idx = threadIdx.x;
	
	switch(idx%val1){
	    case 0: barrier(d_in,val2);break;
	    case 1: barrier(d_in,val2);break;
	    case 2: barrier(d_in,val2);break;
	    case 3: barrier(d_in,val2);break;
	    case 4: barrier(d_in,val2);break;
	    case 5: barrier(d_in,val2);break;
	    case 6: barrier(d_in,val2);break;
	    case 7: barrier(d_in,val2);break;
	    case 8: barrier(d_in,val2);break;
	    case 9: barrier(d_in,val2);break;
	    case 10: barrier(d_in,val2);break;
	    case 11: barrier(d_in,val2);break;
	    case 12: barrier(d_in,val2);break;
	    case 13: barrier(d_in,val2);break;
	    case 14: barrier(d_in,val2);break;
	    case 15: barrier(d_in,val2);break;
	    case 16: barrier(d_in,val2);break;
	    case 17: barrier(d_in,val2);break;
	    case 18: barrier(d_in,val2);break;
	    case 19: barrier(d_in,val2);break;
	    case 20: barrier(d_in,val2);break;
	    case 21: barrier(d_in,val2);break;
	    case 22: barrier(d_in,val2);break;
	    case 23: barrier(d_in,val2);break;
	    case 24: barrier(d_in,val2);break;
	    case 25: barrier(d_in,val2);break;
	    case 26: barrier(d_in,val2);break;
	    case 27: barrier(d_in,val2);break;
	    case 28: barrier(d_in,val2);break;
	    case 29: barrier(d_in,val2);break;
	    case 30: barrier(d_in,val2);break;
	    case 31: barrier(d_in,val2);break;
	}
	__syncthreads();
}




int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 1024;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	
	
	for(int v1=31;v1>=1;v1-=2){
		for(int v2=10000;v2>=1000;v2-=400){
		
			struct timeval  tv1, tv2;
			gettimeofday(&tv1, NULL);

			for(long long int j=0;j<3000000;j++){
				five<<<1, ARRAY_SIZE>>>(d_out, d_in,v1,v2);
				hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
				hipDeviceSynchronize();
			}
			
			gettimeofday(&tv2, NULL);
			printf ("%d %d %f\n",v1,v2,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
		}
	}
	// copy back the result array to the CPU
	

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
		

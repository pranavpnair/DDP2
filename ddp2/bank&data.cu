#include<stdio.h>
#include<hip/hip_runtime.h>
#include <sys/time.h>


// Device code
__global__ void kernel(int* d_A, int pitch,int height,int width)
{

    for (int c = 0; c < height; ++c) {
        for (int r = 0; r < width; ++r) {
             int* row = (int*)((char*)d_A + r * pitch);
             row[c] = row[c]*row[c];
        }
    }
}




//Host Code
int main()
{
    int* d_A;
    size_t pitch;
    int *A;
    int height,width;

    height = width = 32;
    int rows = height;
    int cols = width;
    A = (int *)malloc(rows*cols*sizeof(int));
    for (int i = 0; i < rows*cols; i++) A[i] = i;
    hipMallocPitch((void**)&d_A, &pitch, width * sizeof(int), height);
    hipMemcpy2D(d_A, pitch, A, sizeof(int)*cols, sizeof(int)*cols, rows, hipMemcpyHostToDevice);
	for(int v1=29;v1>=1;v1-=2){
	for(int v2=10000;v2>=100;v2-=100){
    	struct timeval  tv1, tv2;
		gettimeofday(&tv1, NULL);
		for(int j=0;j<5000000;j++)
    		kernel<<<100, 32>>>(d_A, pitch,v2,v1);
    	gettimeofday(&tv2, NULL);

		printf ("%d %d %f\n",v1,v2,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
    	
    hipDeviceSynchronize();
	}}
   // for(int i=0;i<rows*cols;i++)
   //     printf("%d %d\n",A[i],d_A[i]);
    return 0;
}

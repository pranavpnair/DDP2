#include <stdio.h>
#include <hip/hip_runtime.h>
#include<sys/time.h>

__global__ void seventeen(unsigned *vector, unsigned vectorsize,int v2){
	unsigned id;
	id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id<vectorsize)   
		vector[id]++;  
	__syncthreads(); 
     for(int g=1;g<=1024;g+=v2){
    	vector[g]++;
    }
}



#define BLOCKSIZE 1024
#define ARRAY_SIZE 1024

int main(int nn, char *str[]) {
    unsigned long long N=1024;
   for(int v1=32;v1>=1;v1-=3){
		for(int j=1024;j>=1;j-=100){
			struct timeval  tv1, tv2;
			gettimeofday(&tv1, NULL);
			unsigned *vector, *hvector;
			unsigned vec[ARRAY_SIZE*j];
			for (int i = 0; i < 1024*j; i++) {
				vec[i] = i;
			}
			hipMalloc(&vector, N * sizeof(unsigned));
			hipMemcpy(vector, vec, N * sizeof(unsigned), hipMemcpyHostToDevice);
			hvector = (unsigned *)malloc(N * sizeof(unsigned));
			unsigned nblocks = ceil((float)N / 1024);
		//    printf("nblocks = %d\n", nblocks);
			seventeen<<<100, 1024>>>(vector, N,v1);
			hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);

			gettimeofday(&tv2, NULL);
			
				printf ("%d %d %f\n",v1,j,
					 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
					 (double) (tv2.tv_sec - tv1.tv_sec));
			hipDeviceSynchronize();		
		}
	}
    return 0;
}

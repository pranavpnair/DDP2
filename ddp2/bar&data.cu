#include <stdio.h>
#include <hip/hip_runtime.h>
#include<sys/time.h>

__global__ void seventeen(unsigned *vector, unsigned vectorsize,int v2){
	unsigned id;
	id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id<vectorsize)   
		vector[id]++;  
	__syncthreads();
	 int flag=0;  
     for(int g=1;g<=v2;g++){
    	if(g%2==0)
    		flag++;
    	else
    		flag--;
    	__syncthreads();//barrier here
 
    }
}



#define BLOCKSIZE 1024
#define ARRAY_SIZE 1024

int main(int nn, char *str[]) {
    unsigned long long N=1024;
   for(int v1=10000;v1>=100;v1-=300){
		for(int j=1024;j>=1;j-=10){
			struct timeval  tv1, tv2;
			gettimeofday(&tv1, NULL);
			unsigned *vector, *hvector;
			unsigned vec[ARRAY_SIZE*j];
			for (int i = 0; i < 1024*j; i++) {
				vec[i] = i;
			}
			hipMalloc(&vector, N * sizeof(unsigned));
			hipMemcpy(vector, vec, N * sizeof(unsigned), hipMemcpyHostToDevice);
			hvector = (unsigned *)malloc(N * sizeof(unsigned));
			unsigned nblocks = ceil((float)N / 1024);
		//    printf("nblocks = %d\n", nblocks);
			seventeen<<<100, 1024>>>(vector, N,v1);
			hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);

			gettimeofday(&tv2, NULL);
			
				printf ("%d %d %f\n",j,v1,
					 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
					 (double) (tv2.tv_sec - tv1.tv_sec));
			hipDeviceSynchronize();		
		}
	}
    return 0;
}

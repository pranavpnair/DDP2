#include<stdio.h>
#include<hip/hip_runtime.h>
#include <sys/time.h>


// Device code
__device__ void kernel(int* d_A, int pitch,int height,int width)
{
    for (int c = 0; c < height; ++c) {
        for (int r = 0; r < width; ++r) {
             int* row = (int*)((char*)d_A + r * pitch);
             row[c] = row[c]*row[c];
        }
    }
}



__global__ void three(int * d_out, int pitch,int height, int width,int val1,int val2){
	
	int idx = threadIdx.x;
	
	switch(idx%val1){
	    case 0: kernel(d_out,pitch,height,val2);break;
	    case 1: kernel(d_out,pitch,height,val2);break;
	    case 2: kernel(d_out,pitch,height,val2);break;
	    case 3: kernel(d_out,pitch,height,val2);break;
	    case 4: kernel(d_out,pitch,height,val2);break;
	    case 5: kernel(d_out,pitch,height,val2);break;
	    case 6: kernel(d_out,pitch,height,val2);break;
	    case 7: kernel(d_out,pitch,height,val2);break;
	    case 8: kernel(d_out,pitch,height,val2);break;
	    case 9: kernel(d_out,pitch,height,val2);break;
	    case 10: kernel(d_out,pitch,height,val2);break;
	    case 11: kernel(d_out,pitch,height,val2);break;
	    case 12: kernel(d_out,pitch,height,val2);break;
	    case 13: kernel(d_out,pitch,height,val2);break;
	    case 14: kernel(d_out,pitch,height,val2);break;
	    case 15: kernel(d_out,pitch,height,val2);break;
	    case 16: kernel(d_out,pitch,height,val2);break;
	    case 17: kernel(d_out,pitch,height,val2);break;
	    case 18: kernel(d_out,pitch,height,val2);break;
	    case 19: kernel(d_out,pitch,height,val2);break;
	    case 20: kernel(d_out,pitch,height,val2);break;
	    case 21: kernel(d_out,pitch,height,val2);break;
	    case 22: kernel(d_out,pitch,height,val2);break;
	    case 23: kernel(d_out,pitch,height,val2);break;
	    case 24: kernel(d_out,pitch,height,val2);break;
	    case 25: kernel(d_out,pitch,height,val2);break;
	    case 26: kernel(d_out,pitch,height,val2);break;
	    case 27: kernel(d_out,pitch,height,val2);break;
	    case 28: kernel(d_out,pitch,height,val2);break;
	    case 29: kernel(d_out,pitch,height,val2);break;
	    case 30: kernel(d_out,pitch,height,val2);break;
	    case 31: kernel(d_out,pitch,height,val2);break;
	}
	__syncthreads();
}


//Host Code
int main()
{
    int* d_A;
    size_t pitch;
    int *A;
    int height,width;

    height = width = 32;
    int rows = height;
    int cols = width;
    A = (int *)malloc(rows*cols*sizeof(int));
    for (int i = 0; i < rows*cols; i++) A[i] = i;
    hipMallocPitch((void**)&d_A, &pitch, width * sizeof(int), height);
    hipMemcpy2D(d_A, pitch, A, sizeof(int)*cols, sizeof(int)*cols, rows, hipMemcpyHostToDevice);
	for(int v1=32;v1>=1;v1-=2){
	for(int v2=32;v2>=1;v2-=2){
    	struct timeval  tv1, tv2;
		gettimeofday(&tv1, NULL);
		for(int j=0;j<1000000;j++)
    		three<<<1, 1024>>>(d_A, pitch,height,width,v1,v2);
    	gettimeofday(&tv2, NULL);

		printf ("%d %d %f\n",v1,v2,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
    	
    hipDeviceSynchronize();
	}}
   // for(int i=0;i<rows*cols;i++)
   //     printf("%d %d\n",A[i],d_A[i]);
    return 0;
}

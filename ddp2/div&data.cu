#include <stdio.h>
#include <hip/hip_runtime.h>
#include<sys/time.h>

__global__ void seventeen(unsigned *vector, unsigned vectorsize,int val1){
	int idx = threadIdx.x;
	unsigned id;
	switch(idx%val1){
	    case 0:     id = blockIdx.x * blockDim.x + threadIdx.x;
    				if(id<vectorsize)   
        			vector[id]++;  break;
	    case 1:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 2:      id = blockIdx.x * blockDim.x + threadIdx.x;
    				if(id<vectorsize)   
        			vector[id]++;  break;
	    case 3:      id = blockIdx.x * blockDim.x + threadIdx.x;
    				if(id<vectorsize)   
        			vector[id]++;  break;
	    case 4:      id = blockIdx.x * blockDim.x + threadIdx.x;
    				if(id<vectorsize)   
        			vector[id]++;  break;
	    case 5:      id = blockIdx.x * blockDim.x + threadIdx.x;
    				if(id<vectorsize)   
        			vector[id]++;  break;
	    case 6:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 7:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 8:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 9:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 10:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 11:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 12:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 13:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 14:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 15:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 16:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 17:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 18:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 19:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 20:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 21:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 22:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 23:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 24:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 25:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 26:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 27:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 28:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 29:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 30:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	    case 31:      id = blockIdx.x * blockDim.x + threadIdx.x;
					if(id<vectorsize)   
						vector[id]++;  break;
	}
	__syncthreads();
}



#define ARRAY_SIZE 1024


int main(int nn, char *str[]) {
    unsigned N=1024;
   for(int v1=32;v1>=1;v1--){
		for(int j=1024;j>=1;j-=10){
			struct timeval  tv1, tv2;
			gettimeofday(&tv1, NULL);
			unsigned *vector, *hvector;
			unsigned vec[ARRAY_SIZE*j];
			for (int i = 0; i < 1024*j; i++) {
				vec[i] = i;
			}
			int N = 1024;
			hipMalloc(&vector, N * sizeof(unsigned));
			hipMemcpy(vector, vec, N * sizeof(unsigned), hipMemcpyHostToDevice);
			hvector = (unsigned *)malloc(N * sizeof(unsigned));
			unsigned nblocks = ceil((float)N / 1024);
		//    printf("nblocks = %d\n", nblocks);
			seventeen<<<100, 1024>>>(vector, N,v1);
			hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);

			gettimeofday(&tv2, NULL);
			
				printf ("%d %d %f\n",v1,j,
					 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
					 (double) (tv2.tv_sec - tv1.tv_sec));
			hipDeviceSynchronize();		
		}
	}
    return 0;
}

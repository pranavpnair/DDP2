#include<stdio.h>
#include<hip/hip_runtime.h>
#include <sys/time.h>


// Device code
__global__ void kernel(int* d_A, int pitch,int height,int width)
{
    for (int c = 0; c < height; ++c) {
        for (int r = 0; r < width; ++r) {
             int* row = (int*)((char*)d_A + r * pitch);
             row[c] = row[c]*row[c];
        }
    }
}

//Host Code
int main()
{
    int* d_A;
    size_t pitch;
    int *A;
    int height,width;
    for(int q=100;q<1000;q+=10){
    height = width = q;
    int rows = height;
    int cols = width;
    A = (int *)malloc(rows*cols*sizeof(int));
    for (int i = 0; i < rows*cols; i++) A[i] = i;
    hipMallocPitch((void**)&d_A, &pitch, width * sizeof(int), height);
    hipMemcpy2D(d_A, pitch, A, sizeof(int)*cols, sizeof(int)*cols, rows, hipMemcpyHostToDevice);

    	struct timeval  tv1, tv2;
		gettimeofday(&tv1, NULL);
		for(int j=0;j<1000000;j++)
    		kernel<<<100, 32>>>(d_A, pitch,height,width);
    	gettimeofday(&tv2, NULL);

		printf ("%d\t%f\n",q,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
    	
    hipDeviceSynchronize();
	}
   // for(int i=0;i<rows*cols;i++)
   //     printf("%d %d\n",A[i],d_A[i]);
    return 0;
}




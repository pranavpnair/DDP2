#include <stdio.h>
#include <hip/hip_runtime.h>
#include<sys/time.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize,int N) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id<vectorsize)   
        vector[id]+=N;  
}

#define BLOCKSIZE 1024


int main(int nn, char *str[]) {
    unsigned long long N = 1024;
    unsigned *vector, *hvector;
    unsigned vec[N];
    for (int i = 0; i < N; i++) {
		vec[i] = i;
	}
	
    hipMalloc(&vector, N * sizeof(unsigned));
    hipMemcpy(vector, vec, N * sizeof(unsigned), hipMemcpyHostToDevice);
    hvector = (unsigned *)malloc(N * sizeof(unsigned));
    unsigned nblocks = ceil((float)N / BLOCKSIZE);
//    printf("nblocks = %d\n", nblocks);

//here we run  the kernel in a loop which runs 1024 times.
	for(int j=1024;j<1000000;j+=10000) {
		struct timeval  tv1, tv2;
		gettimeofday(&tv1, NULL);
    	for(int i=0;i<j;i++)
    	    dkernel<<<nblocks, BLOCKSIZE>>>(vector, N,i);
    	gettimeofday(&tv2, NULL);

		printf ("%d\t%f\n",j,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
   	}
    hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
    /*for (unsigned ii = 0; ii < N; ++ii) {
    printf("%4d ", hvector[ii]);
    }*/
    return 0;
}

#include <stdio.h>
#include <hip/hip_runtime.h>
#include<sys/time.h>

__global__ void dkernel(unsigned *vector, unsigned vectorsize) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id<vectorsize)   
        vector[id]++;  
}

#define BLOCKSIZE 1024


int main(int nn, char *str[]) {
    unsigned long long N=1024;
    for(int j=1;j<1024;j+=10){
    struct timeval  tv1, tv2;
	gettimeofday(&tv1, NULL);
    unsigned *vector, *hvector;
    unsigned vec[N*j];
    for (int i = 0; i < N*j; i++) {
		vec[i] = i;
	}
	
    hipMalloc(&vector, N * sizeof(unsigned));
    hipMemcpy(vector, vec, N * sizeof(unsigned), hipMemcpyHostToDevice);
    hvector = (unsigned *)malloc(N * sizeof(unsigned));
    unsigned nblocks = ceil((float)N / BLOCKSIZE);
//    printf("nblocks = %d\n", nblocks);
    dkernel<<<nblocks, BLOCKSIZE>>>(vector, N);
    hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
    
    gettimeofday(&tv2, NULL);

		printf ("%d\t%f\n",j,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
    }
    return 0;
}

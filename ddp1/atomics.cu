

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>


__global__ void square( int * d_in,int n){
    int totalSum;
	if (threadIdx.x == 0) totalSum = 0;
    __syncthreads();

    int localVal = d_in[threadIdx.x];
    for(int i=0;i<n;i++)
    	atomicAdd(&totalSum, 1);
    __syncthreads();
}

int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	int h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = i;
	}
	int * d_in;
	hipMalloc((void**) &d_in, ARRAY_BYTES);
//	cudaMalloc((void*) &totalSum, sizeof(float));
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	
	
	for(int i=100;i<1000;i+=10){
		struct timeval  tv1, tv2;
		gettimeofday(&tv1, NULL);
		for(int j=0;j<1000000;j++)
			square<<<1, 64>>>(d_in,i);
		gettimeofday(&tv2, NULL);

		printf ("%d\t%f\n",i,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
	}
//	cudaMemcpy(ans, totalSum, sizeof(float), cudaMemcpyDeviceToHost);
//    printf("%f\n",ans);
	hipFree(d_in);

	return 0;
}

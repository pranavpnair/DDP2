
#include <hip/hip_runtime.h>
#include <stdio.h>
 
__global__ void kernel(int *d_A, size_t pitch, int rows, int cols){
  //compute the row
  int r = blockIdx.y*blockDim.y+threadIdx.y;
  //compute the column
  int c = blockIdx.x*blockDim.x+threadIdx.x;
 
  if((r < rows) && (c < cols)){
  //   // update the pointer to point to the beginning of the row
    int *Row = (int*)((char*)d_A + r*pitch);
    int elem = Row[c];
    printf("%d ", elem);
  }
}
 
int main(){
 
  int *d_A, *A;
  size_t pitch;
  int rows = 4;
  int cols = 4;
  A = (int *)malloc(rows*cols*sizeof(int));
  for (int i = 0; i < rows*cols; i++) A[i] = i;
  hipMallocPitch((void**)&d_A, &pitch, sizeof(int)*cols, rows);
  hipMemcpy2D(d_A, pitch, A, sizeof(int)*cols, sizeof(int)*cols, rows, hipMemcpyHostToDevice);
  dim3 block(16,16);
  dim3 grid(1,1);
  kernel<<<grid,block>>>(d_A, pitch, rows, cols);
  hipDeviceSynchronize();
  printf("\nDone!\n");
  return 0;
}

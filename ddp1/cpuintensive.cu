
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX 500


__global__ void dkernel(int *res,int n,int *res_size) {
    // Initialize result
    res[0] = 1;
//    int res_size = 1;
 
    for (int x=2; x<=n; x++){
        int carry = 0;  // Initialize carry
        for (int i=0; i<*res_size; i++)
        {
            int prod = res[i] * x + carry;
            res[i] = prod % 10; 
            carry  = prod/10;    
        }
     
        while (carry)
        {
            res[*res_size] = carry%10;
            carry = carry/10;
            (*res_size)++;
        }
    
    }
}


void soln(int *res,int n,int *res_size){
    // Initialize result
    res[0] = 1;
//    int res_size = 1;
 	int x;
    for (x=2; x<=n; x++){
        int carry = 0,i;  // Initialize carry
        for (i=0; i<*res_size; i++)
        {
            int prod = res[i] * x + carry;
            res[i] = prod % 10; 
            carry  = prod/10;    
        }
     
        while (carry)
        {
            res[*res_size] = carry%10;
            carry = carry/10;
            (*res_size)++;
        }
    
    }
}


int main(int nn, char *str[]) {
    int N = atoi(str[1]);
    int *res, *hres;
    int res_size=1;
    
    
    hipMalloc(&res, MAX * sizeof(int));
    hres = (int*)malloc(MAX*sizeof(int));
//    soln(hres,N,&res_size);
    dkernel<<<1,1>>>(res,N,&res_size);
    hipMemcpy(hres, res, MAX * sizeof(int), hipMemcpyDeviceToHost);
    //    cout << "Factorial of given number is \n";
    int i;
    for (i=res_size-1; i>=0; i--)
        printf("%d",hres[i]);
    printf("\n");
    return 0;
}

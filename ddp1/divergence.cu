
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<sys/time.h>


__global__ void square(float * d_out, float * d_in,int mod){
	
	int idx = threadIdx.x;
	float f = d_in[idx];
	switch(idx%32){
	    case 0: d_out[idx] = f*f;break;
	    case 1: d_out[idx] = f*f;break;
	    case 2: d_out[idx] = f*f;break;
	    case 3: d_out[idx] = f*f;break;
	    case 4: d_out[idx] = f*f;break;
	    case 5: d_out[idx] = f*f;break;
	    case 6: d_out[idx] = f*f;break;
	    case 7: d_out[idx] = f*f;break;
	    case 8: d_out[idx] = f*f;break;
	    case 9: d_out[idx] = f*f;break;
	    case 10: d_out[idx] = f*f;break;
	    case 11: d_out[idx] = f*f;break;
	    case 12: d_out[idx] = f*f;break;
	    case 13: d_out[idx] = f*f;break;
	    case 14: d_out[idx] = f*f;break;
	    case 15: d_out[idx] = f*f;break;
	    case 16: d_out[idx] = f*f;break;
	    case 17: d_out[idx] = f*f;break;
	    case 18: d_out[idx] = f*f;break;
	    case 19: d_out[idx] = f*f;break;
	    case 20: d_out[idx] = f*f;break;
	    case 21: d_out[idx] = f*f;break;
	    case 22: d_out[idx] = f*f;break;
	    case 23: d_out[idx] = f*f;break;
	    case 24: d_out[idx] = f*f;break;
	    case 25: d_out[idx] = f*f;break;
	    case 26: d_out[idx] = f*f;break;
	    case 27: d_out[idx] = f*f;break;
	    case 28: d_out[idx] = f*f;break;
	    case 29: d_out[idx] = f*f;break;
	    case 30: d_out[idx] = f*f;break;
	    case 31: d_out[idx] = f*f;break;
	}
}

int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	
	

	for(int i=1;i<100;i++){
		
		struct timeval  tv1, tv2;
		gettimeofday(&tv1, NULL);

		
		for(long long int j=0;j<10000000;j++)
			square<<<i, ARRAY_SIZE>>>(d_out, d_in,32);
		
		gettimeofday(&tv2, NULL);
		printf ("%d\t%f\n",i,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
	}

	
	
	

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}

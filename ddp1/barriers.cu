#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize,int i) {
 unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    vector[id] = id;
    for(int g=1;g<=i;g++)
    	__syncthreads();//barrier here
 
    
}
#define BLOCKSIZE 1024


int main(int nn, char *str[]) {
    unsigned N = 1024;
    unsigned *vector, *hvector;
    hipMalloc(&vector, N * sizeof(unsigned));
    hvector = (unsigned *)malloc(N * sizeof(unsigned));
    unsigned nblocks = ceil((float)N / BLOCKSIZE);
    printf("nblocks = %d\n", nblocks);
     for(int i=1;i<=1000;i+=200){
    	struct timeval  tv1, tv2;
		gettimeofday(&tv1, NULL);
		for(long int j=0;j<10000000;j++)
    		dkernel<<<i, i>>>(vector, N,i);
    gettimeofday(&tv2, NULL);

		printf ("%d\t%f\n",i,
			 (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
			 (double) (tv2.tv_sec - tv1.tv_sec));
    	
    }
    hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
 /*   for (unsigned ii = 0; ii < N; ++ii) {
    printf("%4d ", hvector[ii]);
    }*/
    return 0;
}
